#include "hip/hip_runtime.h"
#include "../carver/carver.h"
#include "cuda_energy.h"
#include "cuda_image.h"

#include <cassert>
#include <algorithm>

// Cuda does not allow calling any functions that are not annotated __global__
// And functions within a class cannot have that annotation because they're implicitly outside of the global namespac.e
// Therefore, shadowing key features of energy in order to get it to work.

// support cuda api: turn this object into a struct.
// all relevant private fields will be exposed.

/**
 * Get the gradient energy difference between two pixels.
 *
 * @param p1 First pixel to consider.
 * @param p2 Second pixel to consider.
 * @param retval Pointer to integer to place energy in.
 */
__global__ void gradient_energy(hpimage::pixel p1, hpimage::pixel p2, uint32_t *retval) {

}

/**
 * Update the specified row and column of a cuda energy to have its basic energy
 * Does not consider neighbor energy.
 *
 * @param c_energy Cuda energy struct containing energy matrix.
 * @param col Col to consider.
 * @param row Row to consider.
 */
__global__ void pixel_energy(hpc_cuda::CudaEnergyStruct c_energy,
                             hpc_cuda::CudaImageStruct c_image, uint32_t col, uint32_t row) {
        
}


/**
 * Given an energy matrix, compute the minimum energy of col considering previous neighbor's energies.
 *
 * @param energy Energy matrix to use.
 * @param col Column to start from. Must be greater than zero, because we're considering backwards neighbor energies.
 */
__global__ void horiz_energy_neighbor(hpc_cuda::CudaEnergyStruct c_energy,
                                      hpc_cuda::CudaImageStruct c_image, uint32_t col) {
    assert(col > 0 && col < c_energy.current_cols);
    assert(c_energy.current_rows > 0);

    int start = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int row = start; row < c_energy.current_rows; row += stride) {
        // Need to get local energy of (col, row).
        // Need static function to do this -- helper must be declared __global__.
        
        // Get the neighbor energies.
        uint32_t left_col = col - 1;

        // While we allow wrapping for calculating basic energies, there is no wrapping in seams.
        // Therefore, each pixel is allowed only to consider the neighbors they have.
        int64_t min_energy = -1;
        if (row > 0) {
            uint32_t top_energy = c_energy.energy[(row - 1) * c_energy.base_cols + left_col];
            min_energy = top_energy;
        }

        uint32_t middle_energy = c_energy.energy[row * c_energy.base_cols + left_col];
        if (min_energy == -1 || middle_energy < min_energy) {
            min_energy = middle_energy;
        }

        if (row + 1 < c_energy.current_rows) {
            uint32_t bottom_energy = c_energy.energy[(row + 1) * c_energy.base_cols + left_col];
            if (bottom_energy < min_energy) {
                min_energy = bottom_energy;
            }
        }

        // Sum the local energy of (col, row) and the minimum neighbor energy.
        // Place this in here.
    }
}

/**
 * Given an energy matrix, compute the minimum energy of row considering preceeding row energies.
 * @param energy Energy matrix to use.
 * @param row Row to start from. Must be greater than zero -- considering predecessor energy.
 */
__global__ void vert_energy_neighbor(uint32_t *energy, uint32_t row, uint32_t rows, uint32_t cols) {
    assert(row > 0);
    assert(cols > 0);

    int start = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = start; i < cols; i += stride) {
    }
}

namespace carver {

// ***** HORIZONTAL SEAM CALCULATORS ***** //

void Carver::horiz_energy() {
    for (auto row = 0; row < energy->rows(); ++row) {
        energy->set_energy(0, row, pixel_energy(0, row));
    }

    // Now set energy to minimum of three neighbors.
    for (auto col = 1; col < energy->cols(); ++col) {
        horiz_energy_neighbor<<<10, 1024>>>(((hpc_cuda::CudaEnergy *) energy)->to_struct(),
                                            ((hpc_cuda::CudaImage *) image)->to_struct(), col);

        // Within a row, we're good.
        for (auto row = 0; row < energy->rows(); ++row) {
            // No wrapping
            auto neighbor_energies = energy->get_left_predecessors(col, row);

            // Energy = local energy + min(neighbors)
            uint32_t local_energy = pixel_energy(col, row);
            local_energy += *std::min_element(neighbor_energies.begin(), neighbor_energies.end());
            energy->set_energy(col, row, local_energy);
        }
    }
}

std::vector<uint32_t> Carver::min_horiz_seam() {
    // Now, prime the reverse traversal with the minimum above energy->
    uint32_t back_col = energy->cols() - 1;
    auto seam = std::vector<uint32_t>{};

    // Default: row 0 of the last column contains the minimum energy->
    // Invariant: there will be at least two rows to consider.
    uint32_t min_row = 0;
    uint32_t min_energy = energy->get_energy(back_col, 0);

    for (auto row = 1; row < energy->rows(); ++row) {
        uint32_t current_energy = energy->get_energy(back_col, row);
        if (current_energy < min_energy) {
            min_row = row;
            min_energy = current_energy;
        }
    }
    seam.push_back(min_row);

    // Find the rest of the seam, using only the three predecessors of each node.
    // Using wider signed form to prevent underflow
    for (int64_t col = back_col - 1; col >= 0; --col) {
        // Get the previous index from which to grab neighbors.
        auto row = seam.back();
        min_row = row;
        min_energy = energy->get_energy(col, min_row);
        // Check if the upper or lower neighbors are actually better choices.
        if (row > 0 && min_energy > energy->get_energy(col, row - 1)) {
            min_row = row - 1;
            min_energy = energy->get_energy(col, row - 1);
        }
        if (row + 1 < energy->rows() && min_energy > energy->get_energy(col, row + 1)) {
            min_row = row + 1;
        }
        seam.push_back(min_row);
    }

    // Finally, reverse seam so that it goes in the natural rear-forward order.
    std::reverse(seam.begin(), seam.end());
    return seam;
}


// ***** VERTICAL SEAM CALCULATORS ***** //

void Carver::vert_energy() {
    // Vertical seam direction: top to bottom
    // Prime memo structure with base energies of first pixel row.
    for (auto col = 0; col < energy->cols(); ++col) {
        energy->set_energy(col, 0, pixel_energy(col, 0));
    }

    // This is one of the larger opportunities for parallelism.
    // Set energy to minimum of three above neighbors.
    for (auto row = 1; row < energy->rows(); ++row) {
        for (auto col = 0; col < energy->cols(); ++col) {
            // Note: no wrapping in seams!
            auto neighbor_energies = energy->get_top_predecessors(col, row);

            // energy = local energy + min(neighbors)
            uint32_t local_energy = pixel_energy(col, row);
            local_energy += *std::min_element(neighbor_energies.begin(), neighbor_energies.end());
            energy->set_energy(col, row, local_energy);
        }
    }
}

std::vector<uint32_t> Carver::min_vert_seam() {
    uint32_t bottom_row = energy->rows() - 1;
    auto seam = std::vector<uint32_t>{};

    // Default: row 0 of the last column contains the minimum energy->
    // Invariant: there will be at least two rows to consider.
    uint32_t min_col = 0;
    uint32_t min_energy = energy->get_energy(0, bottom_row);

    for (auto col = 1; col < energy->cols(); ++col) {
        uint32_t current_energy = energy->get_energy(col, bottom_row);
        if (current_energy < min_energy) {
            min_col = col;
            min_energy = current_energy;
        }
    }

    seam.push_back(min_col);

    // Find the rest of the seam, using only the three predecessors of each node.
    // Using wider signed form to prevent underflow
    for (int64_t row = bottom_row - 1; row >= 0; --row) {
        // Get the previous index from which to grab neighbors
        auto col = seam.back();
        min_col = col;
        min_energy = energy->get_energy(min_col, row);
        // Check if the upper or lower neighbors are actually better choices.
        if (col > 0 && min_energy > energy->get_energy(col - 1, row)) {
            min_col = col - 1;
            min_energy = energy->get_energy(col - 1, row);
        }
        if (col + 1 < energy->cols() && min_energy > energy->get_energy(col + 1, row)) {
            min_col = col + 1;
        }
        seam.push_back(min_col);
    }

    // Reverse the seam so traversal happens in expected direction.
    std::reverse(seam.begin(), seam.end());
    return seam;
}


// ***** SEAM REMOVERS ***** //

void Carver::remove_horiz_seam(std::vector<uint32_t> &seam) {
    // Must be exactly one row to remove from each column.
    assert(seam.size() == image->cols());

    for (auto col = 0; col < image->cols(); ++col) {
        auto index = seam[col];
        assert(index < image->rows());

        // Shift all pixels below this up one.
        for (auto row = index; row < image->rows() - 1; ++row) {
            hpimage::pixel below = image->get_pixel(col, row + 1);
            image->set_pixel(col, row, below);
        }
    }
    // Finally, cut the last row from the pixel.
    energy->cut_row();
    image->cut_row();
}

void Carver::remove_vert_seam(std::vector<uint32_t> &seam) {
    // Must be exactly one column to remove from each row.
    assert(seam.size() == image->rows());

    // Shift every pixel after a given image over.
    // Then reduce image size by one.
    for (auto row = 0; row < image->rows(); ++row) {
        auto index = seam[row];
        assert(index < image->cols());

        // Shift all pixels after this one back
        for (auto col = index; col < image->cols() - 1; ++col) {
            hpimage::pixel next = image->get_pixel(col + 1, row);
            image->set_pixel(col, row, next);
        }
    }
    // Finally, with all pixels shifted over, time to trim the image!
    energy->cut_col();
    image->cut_col();
}
} // namespace carver
