#include "cuda_image.h"

using namespace hpc_cuda;

hpimage::pixel *CudaImage::alloc(int size) {
    hpimage::pixel *pixel;
    hipMallocManaged(&pixel, size);
    return pixel;
}

CudaImage::~CudaImage() {
    hipFree(pixels);
}