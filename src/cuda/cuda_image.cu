#include "cuda_image.h"
#include <cassert>

using namespace hpc_cuda;

hpimage::pixel *CudaImage::alloc(int size) {
    hpimage::pixel *pixel;
    hipMallocManaged(&pixel, size);
    return pixel;
}

CudaImage::~CudaImage() {
    // Ensure all computation is finished prior to freeing.
    hipDeviceSynchronize();
    assert(hipFree(pixels) == 0);
}